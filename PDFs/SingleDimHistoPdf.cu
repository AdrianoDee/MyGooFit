#include "hip/hip_runtime.h"
#include "SingleDimHistoPdf.hh"

MEM_CONSTANT fptype* dev_base_sidimhisto[100]; // Multiple histograms for the case of multiple PDFs



  //  EXEC_TARGET fptype interpolateArrays (fptype* xArray, fptype* yArray, int intOrder,fptype xvalue)
  //  {
   //
  //     printf("Bin histo pdf 3.0 = %.3f %d \n",xvalue,intOrder);
   //
   //
  //     fptype den,dif,dift,ho,hp,w,y,dy;
  //     fptype coeffC[20], coeffD[20];
   //
  //     dif = fabs(xvalue-xArray[0]) ;
   //
  //     int ns=1;
   //
  //     for(int intexInter =1 ; intexInter<=intOrder+1 ; ++intexInter)
  //     {
  //       dift=fabs(xvalue-xArray[intexInter-1]);
  //       if (dift<dif)
  //       {
  //          ns = intexInter;
  //          dif = dift ;
  //       }
   //
  //       coeffC[intexInter] = yArray[intexInter-1];
  //       coeffD[intexInter] = yArray[intexInter-1];
   //
  //       printf("Bin histo pdf 3 = %.3f %d %.3f %.3f %.3f %.3f \n",xvalue,intexInter,dift,dif,xArray[intexInter-1],coeffC[intexInter],coeffD[intexInter]);
   //
   //
  //     }
   //
  //     y=yArray[--ns] ;
   //
  //     for(int m=1 ; m<intOrder+1; m++)
  //     {
  //       for(int intexInter=1 ; intexInter<=intOrder+1-m ; intexInter++)
  //       {
  //         ho=xArray[intexInter-1]-xvalue ;
  //         hp=xArray[intexInter-1+m]-xvalue ;
  //         w=coeffC[intexInter+1]-coeffD[intexInter] ;
  //         den=ho-hp ;
  //         if (den==0.)
  //         {
  //           return 0. ;
  //         }
  //         den = w/den ;
  //         coeffD[intexInter]=hp*den ;
  //         coeffC[intexInter]=ho*den;
  //         }
  //         dy = (2*ns)<(intOrder+1-m) ? coeffC[ns+1] : coeffD[ns--] ;
  //         y += dy ;
   //
  //         printf("Bin histo pdf 4 = %.3f %.3f %.3f %.3f %.3f %.3f\n",xvalue,ho,hp,w,den,dy);
   //
   //
  //       }
   //
  //       return y;
   //
  //  }


  //  EXEC_TARGET fptype interSingleDimension (int localNumBins, fptype step, fptype lowerBound, fptype xval, int intOrder,fptype* histogram)
  //  {
   //
  //    int localBin    = (int) FLOOR((xval-lowerBound)/step); // Int_t fbinC = dim.getBin(*binning) ;
   //
  //    fptype binCenter = (fptype)localBin*step+lowerBound-0.5*step;
  //    fptype upperBound   = lowerBound + step*localNumBins;
   //
  //    int binOffset = (xval<binCenter)? 1 : 0;
  //    int fbinLo  = localBin - intOrder/2 - binOffset;//Int_t fbinLo = fbinC-intOrder/2 - ((xval<binning->binCenter(fbinC))?1:0) ;
   //
  //    fptype xarr[20];
  //    fptype yarr[20];
   //
  //    printf("Bin histo pdf 1 = %.3f %d %.3f %d %.3f %.3f %.3f %d \n",xval,localBin,binCenter,fbinLo,lowerBound,step,upperBound,intOrder);
   //
  //    for (int index=fbinLo ; index<=intOrder+fbinLo ; ++index)
  //    {
  //      int ibin ;
  //      if (index>=0 && index<localNumBins) {
  //        ibin = index;
  //        xarr[index-fbinLo] = lowerBound+ibin*step-step*0.5;
  //        yarr[index-fbinLo] = histogram[ibin];
  //        printf("Bin histo pdf 2 = %.3f %d %d %d %d %.3f %.3f \n",xval,localBin,index,ibin,localNumBins,xarr[index-fbinLo],histogram[ibin]);
  //      } else if (index>=localNumBins) {
  //       //  ibin = 2*localNumBins-index-1 ;
  //        printf("Over binning 2 \n");
  //        xarr[index-fbinLo] = upperBound+(1e-10)*(index-localNumBins+1);
  //        yarr[index-fbinLo] = 0.0 ;
  //      } else {
  //        printf("Under binning 2 \n");
  //        ibin = -index - 1 ;
  //        xarr[index-fbinLo] = lowerBound-ibin*(1e-10);
  //        yarr[index-fbinLo] = 0.0 ;
  //      }
  //    }
   //
  //    fptype ret = interpolateArrays(xarr,yarr,intOrder+1,xval);
   //
  //    return ret;
   //
  //  }

  //  EXEC_TARGET fptype interSingleDimensionMulti (int otherBin, int localNumBins,int otherNumBins, fptype step, fptype lowerBound, fptype xval, int intOrder,fptype* histogram)
  //  {
   //
  //    int localBin    = (int) FLOOR((xval-lowerBound)/step); // Int_t fbinC = dim.getBin(*binning) ;
   //
  //    fptype binCenter = (fptype)localBin*step+lowerBound-0.5*step;
  //    fptype upperBound   = lowerBound + step*localNumBins;
   //
  //    int binOffset = (xval<binCenter)? 1 : 0;
  //    int fbinLo  = localBin - intOrder/2 - binOffset;//Int_t fbinLo = fbinC-intOrder/2 - ((xval<binning->binCenter(fbinC))?1:0) ;
   //
  //    fptype xarr[20];
  //    fptype yarr[20];
   //
  //    printf("Bin histo Multi pdf 1 = %.3f %d %.3f %d %.3f %.3f %.3f %d \n",xval,localBin,binCenter,fbinLo,lowerBound,step,upperBound,intOrder);
   //
  //    for (int index=fbinLo ; index<=intOrder+fbinLo ; ++index)
  //    {
  //      int ibin ;
  //      int globalBin;
   //
  //      if (index>=0 && index<localNumBins) {
  //        ibin = index;
  //        xarr[index-fbinLo] = lowerBound+ibin*step-step*0.5;
  //        globalBin = otherBin + otherNumBins*ibin;
  //        yarr[index-fbinLo] = histogram[globalBin];
  //        printf("Bin histo Multi pdf 2 = %.3f %d %d %d %d %.3f %.3f \n",xval,localBin,index,ibin,localNumBins,xarr[index-fbinLo],histogram[ibin]);
  //      } else if (index>=localNumBins) {
  //       //  ibin = 2*localNumBins-index-1 ;
  //        xarr[index-fbinLo] = upperBound+(1e-10)*(index-localNumBins+1);
  //        yarr[index-fbinLo] = 0.0 ;
  //      } else {
  //        ibin = -index - 1 ;
  //        xarr[index-fbinLo] = lowerBound-ibin*(1e-10);
  //        yarr[index-fbinLo] = 0.0 ;
  //      }
  //    }
   //
  //    fptype ret = interpolateArrays(xarr,yarr,intOrder+1,xval);
   //
  //    return ret;
   //
  //  }

   EXEC_TARGET fptype device_SiDimHistoPdf (fptype* evt, fptype* p, unsigned int* indices) {
     // Structure is
     // nP totalHistograms interPolationOrder (limit1 step1 bins1) (limit2 step2 bins2) nO o1 o2
     // where limit and step are indices into functorConstants.

     int numVars = (indices[0] - 1) / 3;

     printf("NumVars = %d \n",numVars);

     int globalBin = 0;
     int previousNofBins = 1;
     int myHistogramIndex = indices[1];
     int interpolationOrder = indices[2];

     fptype* myHistogram = dev_base_sidimhisto[myHistogramIndex];

     if(numVars==1)
     {
       printf("NumVars = 1\n");

       int i = 0;

       int localNumBins = indices[3*(i+1) + 1 + 1];
       int lowerBoundIdx   = 2 + 3*i + 1;
       fptype lowerBound   = functorConstants[indices[lowerBoundIdx + 0]];
       fptype step         = functorConstants[indices[lowerBoundIdx + 1]];
       fptype upperBound   = lowerBound + step*localNumBins;

       fptype currVariable = evt[indices[indices[0] + 2]];

       if(currVariable<lowerBound || currVariable >upperBound) return 0.0;

       fptype ret = interSingleDimension(localNumBins, step, lowerBound, currVariable, interpolationOrder, myHistogram);

       return ret;

     }
     else
      return 0.0;

     }

MEM_DEVICE device_function_ptr ptr_to_SiDimHistogram = device_SiDimHistoPdf;

__host__ SingleDimHistoPdf::SingleDimHistoPdf (std::string n,
							 BinnedDataSet* x,
							 std::vector<Variable*> obses, unsigned int interOrder)
  : GooPdf(0, n)
  , numVars(x->numVariables())
{
  if(numVars>2) abortWithCudaPrintFlush(__FILE__, __LINE__, "Only the first two variables will be taken into account !\n");
  if(interOrder>20) abortWithCudaPrintFlush(__FILE__, __LINE__, "Interpolation order must be smaller than 20! \n");

  printf("NumVars Declaration = %d \n",numVars);

  int numConstants = 2*numVars+1;
  registerConstants(numConstants);
  static unsigned int totalHistograms = 0;
  host_constants = new fptype[numConstants];
  totalEvents = 0;

  std::vector<unsigned int> pindices;
  pindices.push_back(totalHistograms);

  host_constants[0] = interOrder;

  pindices.push_back(interOrder);

  int varIndex = 0;
  for (varConstIt var = x->varsBegin(); var != x->varsEnd(); ++var) {
    if (std::find(obses.begin(), obses.end(), *var) != obses.end()) {
      registerObservable(*var);
    }
    else {
      abortWithCudaPrintFlush(__FILE__, __LINE__, "The BinnedDataSet provided variables are different from p.d.f. observables \n");
    }

    pindices.push_back(cIndex + 2*varIndex + 0 + 1); //cIndex is no. of constants index
    pindices.push_back(cIndex + 2*varIndex + 1 + 1);
    pindices.push_back((*var)->numbins);

    // NB, do not put cIndex here, it is accounted for by the offset in MEMCPY_TO_SYMBOL below.
    host_constants[2*varIndex + 0 + 1] = (*var)->lowerlimit;
    host_constants[2*varIndex + 1 + 1] = ((*var)->upperlimit - (*var)->lowerlimit) / (*var)->numbins;
    varIndex++;
  }

  unsigned int numbins = x->getNumBins();
  thrust::host_vector<fptype> host_histogram;
  for (unsigned int i = 0; i < numbins; ++i) {
    fptype curr = x->getBinContent(i);
    host_histogram.push_back(curr);
    totalEvents += curr;
  }
  MEMCPY_TO_SYMBOL(functorConstants, host_constants, numConstants*sizeof(fptype), cIndex*sizeof(fptype), hipMemcpyHostToDevice);

  dev_base_histogram = new thrust::device_vector<fptype>(host_histogram);
  static fptype* dev_address[1];
  dev_address[0] = (&((*dev_base_histogram)[0])).get();
  MEMCPY_TO_SYMBOL(dev_base_sidimhisto, dev_address, sizeof(fptype*), totalHistograms*sizeof(fptype*), hipMemcpyHostToDevice);
  GET_FUNCTION_ADDR(ptr_to_SiDimHistogram);
  initialise(pindices);

  totalHistograms++;
}

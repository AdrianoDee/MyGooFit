#include "hip/hip_runtime.h"
#include "PdfBase.hh"

// This is code that belongs to the PdfBase class, that is, 
// it is common across all implementations. But it calls on device-side
// functions, and due to the nvcc translation-unit limitations, it cannot
// sit in its own object file; it must go in the CUDAglob.cu. So it's
// off on its own in this inline-cuda file, which GooPdf.cu 
// should include. 

#ifdef CUDAPRINT
__host__ void PdfBase::copyParams (const std::vector<double>& pars) const {
  if (host_callnumber < 1) {
    std::cout << "Copying parameters: " << (long long) hipArray << " ";
  }
  for (unsigned int i = 0; i < pars.size(); ++i) {
    host_params[i] = pars[i]; 
    
    if (host_callnumber < 1) {
      std::cout << pars[i] << " ";
    }
    
    if (isnan(host_params[i])) {
      std::cout << " agh, NaN, die " << i << std::endl;
      abortWithCudaPrintFlush(__FILE__, __LINE__, "NaN in parameter"); 
    }
  }
  
  if (host_callnumber < 1) {
    std::cout << std::endl; 
  }
  MEMCPY_TO_SYMBOL(hipArray, host_params, pars.size()*sizeof(fptype), 0, hipMemcpyHostToDevice); 
}
#else 
__host__ void PdfBase::copyParams (const std::vector<double>& pars) const {
  // copyParams method performs eponymous action! 

  for (unsigned int i = 0; i < pars.size(); ++i) {
    host_params[i] = pars[i]; 
    
    if (isnan(host_params[i])) {
      std::cout << " agh, parameter is NaN, die " << i << std::endl;
      abortWithCudaPrintFlush(__FILE__, __LINE__, "NaN in parameter"); 
    }
  }

  MEMCPY_TO_SYMBOL(hipArray, host_params, pars.size()*sizeof(fptype), 0, hipMemcpyHostToDevice); 
}
#endif

__host__ void PdfBase::copyParams () {
  // Copies values of Variable objects
  parCont pars; 
  getParameters(pars); 
  std::vector<double> values; 
  for (parIter v = pars.begin(); v != pars.end(); ++v) {
    int index = (*v)->getIndex(); 
    if (index >= (int) values.size()) values.resize(index + 1);
    values[index] = (*v)->value;
  }
  copyParams(values); 
}

__host__ void PdfBase::copyNormFactors () const {
  MEMCPY_TO_SYMBOL(normalisationFactors, host_normalisation, totalParams*sizeof(fptype), 0, hipMemcpyHostToDevice); 
  SYNCH(); // Ensure normalisation integrals are finished
}

__host__ void PdfBase::initialiseIndices (std::vector<unsigned int> pindices) {
  // Structure of the individual index array: Number of parameters, then the indices
  // requested by the subclass (which will be interpreted by the subclass kernel), 
  // then the number of observables, then the observable indices. Notice that the
  // observable indices are not set until 'setIndices' is called, usually from setData;
  // here we only reserve space for them by setting totalParams. 
  // This is to allow index sharing between PDFs - all the PDFs must be constructed 
  // before we know what observables exist. 

  if (totalParams + pindices.size() >= maxParams) {
    std::cout << "Major problem with pindices size: " << totalParams << " + " << pindices.size() << " >= " << maxParams << std::endl; 
  }

  assert(totalParams + pindices.size() < maxParams); 
  host_indices[totalParams] = pindices.size(); 
  for (int i = 1; i <= host_indices[totalParams]; ++i) {
    host_indices[totalParams+i] = pindices[i-1]; 
  }
  host_indices[totalParams + pindices.size() + 1] = observables.size(); 
  
  parameters = totalParams;
  totalParams += (2 + pindices.size() + observables.size()); 
  /*
  std::cout << "host_indices after " << getName() << " initialisation : ";
  for (int i = 0; i < totalParams; ++i) {
    std::cout << host_indices[i] << " ";
  }
  
  std::cout << " | " 
	    << parameters << " " 
	    << totalParams << " " 
	    << hipArray << " " 
	    << paramIndices << " "
	    << std::endl; 
  */
  MEMCPY_TO_SYMBOL(paramIndices, host_indices, totalParams*sizeof(unsigned int), 0, hipMemcpyHostToDevice); 
}

__host__ void PdfBase::setData (std::vector<std::map<Variable*, fptype> >& data) {
  // Old method retained for backwards compatibility 

  if (dev_event_array) {
    gooFree(dev_event_array);
    dev_event_array = 0; 
  }

  setIndices();
  int dimensions = observables.size();
  numEntries = data.size();
  numEvents = numEntries; 
  
  fptype* host_array = new fptype[data.size()*dimensions];
  for (unsigned int i = 0; i < data.size(); ++i) {
    for (obsIter v = obsBegin(); v != obsEnd(); ++v) {
      assert(data[i].find(*v) != data[i].end()); 
      host_array[i*dimensions + (*v)->index] = data[i][*v]; 
    }
  }

  gooMalloc((void**) &dev_event_array, dimensions*numEntries*sizeof(fptype)); 
  MEMCPY(dev_event_array, host_array, dimensions*numEntries*sizeof(fptype), hipMemcpyHostToDevice);
  MEMCPY_TO_SYMBOL(functorConstants, &numEvents, sizeof(fptype), 0, hipMemcpyHostToDevice); 
  delete[] host_array; 
}

__host__ void PdfBase::recursiveSetIndices () {
  for (unsigned int i = 0; i < components.size(); ++i) {
    components[i]->recursiveSetIndices(); 
  }

  int numParams = host_indices[parameters]; 
  int counter = 0; 
  for (obsIter v = obsBegin(); v != obsEnd(); ++v) {
    host_indices[parameters + 2 + numParams + counter] = (*v)->index; 
    //std::cout << getName() << " set index of " << (*v)->name << " to " << (*v)->index << " " << (parameters + 2 + numParams + counter) << std::endl; 
    counter++; 
  }  
  generateNormRange(); 
}

__host__ void PdfBase::setIndices () {
  int counter = 0; 
  for (obsIter v = obsBegin(); v != obsEnd(); ++v) {
    (*v)->index = counter++; 
  }
  recursiveSetIndices(); 
  MEMCPY_TO_SYMBOL(paramIndices, host_indices, totalParams*sizeof(unsigned int), 0, hipMemcpyHostToDevice); 

  //std::cout << "host_indices after " << getName() << " observable setIndices : ";
  //for (int i = 0; i < totalParams; ++i) {
  //std::cout << host_indices[i] << " ";
  //}
  //std::cout << std::endl; 

}

__host__ void PdfBase::setData (UnbinnedDataSet* data) {
  if (dev_event_array) {
    gooFree(dev_event_array);
    SYNCH();
    dev_event_array = 0; 
  }

  setIndices();
  int dimensions = observables.size();
  numEntries = data->getNumEvents(); 
  numEvents = numEntries; 
  
  fptype* host_array = new fptype[numEntries*dimensions];
  for (int i = 0; i < numEntries; ++i) {
    for (obsIter v = obsBegin(); v != obsEnd(); ++v) {
      fptype currVal = data->getValue((*v), i);
      host_array[i*dimensions + (*v)->index] = currVal; 
    }
  }

  gooMalloc((void**) &dev_event_array, dimensions*numEntries*sizeof(fptype)); 
  MEMCPY(dev_event_array, host_array, dimensions*numEntries*sizeof(fptype), hipMemcpyHostToDevice);
  MEMCPY_TO_SYMBOL(functorConstants, &numEvents, sizeof(fptype), 0, hipMemcpyHostToDevice); 
  delete[] host_array; 
}

__host__ void PdfBase::setData (BinnedDataSet* data) { 
  if (dev_event_array) { 
    gooFree(dev_event_array);
    dev_event_array = 0; 
  }

  setIndices();
  numEvents = 0; 
  numEntries = data->getNumBins(); 
  int dimensions = 2 + observables.size(); // Bin center (x,y, ...), bin value, and bin volume. 
  if (!fitControl->binnedFit()) setFitControl(new BinnedNllFit()); 

  fptype* host_array = new fptype[numEntries*dimensions]; 

  for (unsigned int i = 0; i < numEntries; ++i) {
    for (obsIter v = obsBegin(); v != obsEnd(); ++v) {
      host_array[i*dimensions + (*v)->index] = data->getBinCenter((*v), i); 
    }

    host_array[i*dimensions + observables.size() + 0] = data->getBinContent(i);
    host_array[i*dimensions + observables.size() + 1] = fitControl->binErrors() ? data->getBinError(i) : data->getBinVolume(i); 
    numEvents += data->getBinContent(i);
  }

  gooMalloc((void**) &dev_event_array, dimensions*numEntries*sizeof(fptype)); 
  MEMCPY(dev_event_array, host_array, dimensions*numEntries*sizeof(fptype), hipMemcpyHostToDevice);
  MEMCPY_TO_SYMBOL(functorConstants, &numEvents, sizeof(fptype), 0, hipMemcpyHostToDevice); 
  delete[] host_array; 
}

__host__ void PdfBase::generateNormRange () {
  if (normRanges) gooFree(normRanges);
  gooMalloc((void**) &normRanges, 3*observables.size()*sizeof(fptype));
  
  fptype* host_norms = new fptype[3*observables.size()];
  int counter = 0; // Don't use index in this case to allow for, eg, 
  // a single observable whose index is 1; or two observables with indices
  // 0 and 2. Make one array per functor, as opposed to variable, to make
  // it easy to pass MetricTaker a range without worrying about which parts
  // to use. 
  for (obsIter v = obsBegin(); v != obsEnd(); ++v) {
    host_norms[3*counter+0] = (*v)->lowerlimit;
    host_norms[3*counter+1] = (*v)->upperlimit;
    host_norms[3*counter+2] = integrationBins > 0 ? integrationBins : (*v)->numbins;
    counter++; 
  }

  MEMCPY(normRanges, host_norms, 3*observables.size()*sizeof(fptype), hipMemcpyHostToDevice);
  delete[] host_norms; 
}

void PdfBase::clearCurrentFit () {
  totalParams = 0; 
  gooFree(dev_event_array);
  dev_event_array = 0; 
}

__host__ void PdfBase::printProfileInfo (bool topLevel) {
#ifdef PROFILING
  if (topLevel) {
    hipError_t err = MEMCPY_FROM_SYMBOL(host_timeHist, timeHistogram, 10000*sizeof(fptype), 0);
    if (hipSuccess != err) {
      std::cout << "Error on copying timeHistogram: " << hipGetErrorString(err) << std::endl;
      return;
    }
    
    std::cout << getName() << " : " << getFunctionIndex() << " " << host_timeHist[100*getFunctionIndex() + getParameterIndex()] << std::endl; 
    for (unsigned int i = 0; i < components.size(); ++i) {
      components[i]->printProfileInfo(false); 
    }
  }
#endif
}



gooError gooMalloc (void** target, size_t bytes) {
// Thrust 1.7 will make the use of THRUST_DEVICE_BACKEND an error
#if THRUST_DEVICE_BACKEND==THRUST_DEVICE_BACKEND_OMP || THRUST_DEVICE_SYSTEM==THRUST_DEVICE_BACKEND_OMP
  target[0] = malloc(bytes);
  if (target[0]) return gooSuccess;
  else return gooErrorMemoryAllocation; 
#else
  return (gooError) hipMalloc(target, bytes); 
#endif
}

gooError gooFree (void* ptr) {
// Thrust 1.7 will make the use of THRUST_DEVICE_BACKEND an error
#if THRUST_DEVICE_BACKEND==THRUST_DEVICE_BACKEND_OMP || THRUST_DEVICE_SYSTEM==THRUST_DEVICE_BACKEND_OMP
  free(ptr);
  return gooSuccess;
#else
  return (gooError) hipFree(ptr); 
#endif
}
